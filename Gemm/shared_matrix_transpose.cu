
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_SIZE 32
#define M 3000
#define N 1000

__managed__ int matrix[N][M];
__managed__ int gpu_result[M][N];
__managed__ int cpu_result[M][N];

__global__ void gpu_matrix_transpose(int in[N][M],int out[M][N])
{
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;
    if(x < M&&y < N)
    {
        out[x][y] = in[y][x];
    }
}
__global__ void gpu_shared_matrix_transpose(int in[N][M],int out[M][N])
{
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;
    /// @brief 避免bank冲突
    __shared__ int smem[BLOCK_SIZE + 1][BLOCK_SIZE + 1];
    if(x < M&& y < N)
    {
        smem[threadIdx.y][threadIdx.x] = in[y][x];
    }
    __syncthreads();
    /// @brief 可以将blockDim.y*blockIdx.y看成是block的坐标
    int x1 = threadIdx.x + blockDim.y*blockIdx.y;
    int y1 = threadIdx.y + blockDim.x*blockIdx.x;

    if(x1<N&&y1<M)
    {
        /// 这里得颠倒
        out[y1][x1] = smem[threadIdx.x][threadIdx.y];
    }
}
void cpu_matrix_transpose(int in[N][M],int out[M][N])
{
    for(int y = 0; y < N; y++)
    {
        for(int x = 0; x < M; x++)
        {
            out[x][y] = in[y][x];
        }
    }
}
int main()
{
    for(int y = 0;y < N; y++)
    {
        for(int x = 0; x < M; x++)
        {
            matrix[y][x] = rand()%1024;
        }
    }
    hipEvent_t start,stop_cpu,stop_gpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_cpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start);
    hipEventSynchronize(start);

    dim3 dimGrid((M+BLOCK_SIZE-1)/BLOCK_SIZE,(N+BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);

    for(int i = 0; i < 20; i++)
    {
        gpu_matrix_transpose<<<dimGrid,dimBlock>>>(matrix,gpu_result);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    cpu_matrix_transpose(matrix,cpu_result);

    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);
    
    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);

    bool errors = false;
    for(int y = 0; y<M; y++)
    {
        for (int x = 0; x < N; x++)
        {
            if(fabs(cpu_result[y][x] - gpu_result[y][x]) > (1.0e-10))
            {
                errors = true;
            }
        }
        
    }

    printf("Result: %s\n", errors?"Error":"Pass");
    printf("CPU time: %.2f\nGPU time: %.2f\n", time_cpu, time_gpu/20.0);

    return 0;
}
