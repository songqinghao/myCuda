
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define BLOCK_SIZE 16
#define N 1000;

__global__ void gpu_matrix_mult(int *a,int *b,int *c,int size)
{
    int x = blockDim.x*blockIdx.x+threadIdx.x;
    int y = blockDim.y*blockIdx.y+threadIdx.y;
    int tmp = 0;
    for(int step = 0;step < size; step++)
    {
        tmp+=a[x*size+step]*b[step*size+y];
    }
    c[x*size+tmp] = tmp;
}

void cpu_matrix_mult(int *a,int *b,int *c,int size)
{
    
    for(int x = 0; x < size; x++)
    {
        for(int y = 0; y < size; y++)
        {
            int tmp = 0;
            for(int step = 0;step < size; step++)
            {
                tmp += a[x*size + step]*b[step*size + y];
            }
            c[x * size + y]=tmp;
        }
    }
}
int main()
{
    const int matrix_size = N;
    int memsize = sizeof(int) * matrix_size * matrix_size;
    int *h_a,*h_b,*h_c,*h_cc;
    hipHostMalloc((void**)&h_a,memsize, hipHostMallocDefault);
    hipHostMalloc((void**)&h_b,memsize, hipHostMallocDefault);
    hipHostMalloc((void**)&h_c,memsize, hipHostMallocDefault);
    hipHostMalloc((void**)&h_cc,memsize, hipHostMallocDefault);
    for(int i = 0; i < matrix_size;i++)
    {
        for(int j = 0;j < matrix_size;j++)
        {
            h_a[i * matrix_size + j]=rand() % 1024;
        }
    }
    for(int i = 0; i < matrix_size;i++)
    {
        for(int j = 0;j < matrix_size;j++)
        {
            h_b[i * matrix_size + j]=rand() % 1024;
        }
    }

    int* d_a,*d_b,*d_c;
    hipMalloc((void**)(&d_a),memsize);
    hipMalloc((void**)(&d_b),memsize);
    hipMalloc((void**)(&d_c),memsize);

    hipMemcpy(d_a,h_a,memsize,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,memsize,hipMemcpyHostToDevice);

    unsigned int grid_rows = (matrix_size+BLOCK_SIZE-1)/BLOCK_SIZE;
    unsigned int grid_cols = (matrix_size+BLOCK_SIZE-1)/BLOCK_SIZE;

    dim3 dimGrid(grid_cols,grid_rows);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);/// warp为32,这32个线程共享一个指令
    gpu_matrix_mult<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,matrix_size);
    hipMemcpy(h_c,d_c,memsize,hipMemcpyDeviceToHost);
    cpu_matrix_mult(h_a,h_b,h_cc,matrix_size);
    
    bool errors = false;
    for(int i = 0;i < matrix_size;i++)
    {
        for(int j = 0;j < matrix_size;j++)
        {
            if(fabs(h_cc[i*matrix_size + j]-h_c[i*matrix_size+j])>(1.0e-10))
            {
                errors = true;
                hipHostFree(h_a);
                hipHostFree(h_b);
                hipHostFree(h_c);
                hipHostFree(h_cc);
                hipFree(d_a);
                hipFree(d_b);
                hipFree(d_c);
                printf("info error!\n");
                return;
            }
        }
    }
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}
